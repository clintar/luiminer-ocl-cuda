#include "hip/hip_runtime.h"
/*
	by Pecho <pecho@2ba.su>
	draft version
	based on SPH
*/

//#include "helper.h"

#define SHL(x, n) ((x) << (n))
#define SHR(x, n) ((x) >> (n))

#define CONST_EXP2 q[i+0] + ROTL64(q[i+1], 5) + q[i+2] + ROTL64(q[i+3], 11) + \
  q[i+4] + ROTL64(q[i+5], 27) + q[i+6] + ROTL64(q[i+7], 32) + \
  q[i+8] + ROTL64(q[i+9], 37) + q[i+10] + ROTL64(q[i+11], 43) + \
  q[i+12] + ROTL64(q[i+13], 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

__global__ void kernelBmw512(unsigned char *hashes) {
	int i;
	uint64 BMW_H[16];

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	BMW_H[0] = 0x8081828384858687ULL;
	BMW_H[1] = 0x88898A8B8C8D8E8FULL;
	BMW_H[2] = 0x9091929394959697ULL;
	BMW_H[3] = 0x98999A9B9C9D9E9FULL;
	BMW_H[4] = 0xA0A1A2A3A4A5A6A7ULL; 
	BMW_H[5] = 0xA8A9AAABACADAEAFULL;
	BMW_H[6] = 0xB0B1B2B3B4B5B6B7ULL; 
	BMW_H[7] = 0xB8B9BABBBCBDBEBFULL;
	BMW_H[8] = 0xC0C1C2C3C4C5C6C7ULL; 
	BMW_H[9] = 0xC8C9CACBCCCDCECFULL;
	BMW_H[10] = 0xD0D1D2D3D4D5D6D7ULL; 
	BMW_H[11] = 0xD8D9DADBDCDDDEDFULL;
	BMW_H[12] = 0xE0E1E2E3E4E5E6E7ULL; 
	BMW_H[13] = 0xE8E9EAEBECEDEEEFULL;
	BMW_H[14] = 0xF0F1F2F3F4F5F6F7ULL; 
	BMW_H[15] = 0xF8F9FAFBFCFDFEFFULL;

	uint64 mv[16],q[32];
    uint64 tmp;

	mv[0] = dec64le2(hashes + (tid*64) +   0);
    mv[1] = dec64le2(hashes + (tid*64) +   8);
    mv[2] = dec64le2(hashes + (tid*64) +  16);
    mv[3] = dec64le2(hashes + (tid*64) +  24);
    mv[4] = dec64le2(hashes + (tid*64) +  32);
    mv[5] = dec64le2(hashes + (tid*64) +  40);
    mv[6] = dec64le2(hashes + (tid*64) +  48);
    mv[7] = dec64le2(hashes + (tid*64) +  56);
	mv[8] = 0x80;
	mv[9] = 0;
	mv[10] = 0;
	mv[11] = 0;
	mv[12] = 0;
	mv[13] = 0;
	mv[14] = 0;
	mv[15] = 0x200;

	tmp = (mv[5] ^ BMW_H[5]) - (mv[7] ^ BMW_H[7]) + (mv[10] ^ BMW_H[10]) + (mv[13] ^ BMW_H[13]) + (mv[14] ^ BMW_H[14]);
	q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[1];
	tmp = (mv[6] ^ BMW_H[6]) - (mv[8] ^ BMW_H[8]) + (mv[11] ^ BMW_H[11]) + (mv[14] ^ BMW_H[14]) - (mv[15] ^ BMW_H[15]);
	q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + BMW_H[2];
	tmp = (mv[0] ^ BMW_H[0]) + (mv[7] ^ BMW_H[7]) + (mv[9] ^ BMW_H[9]) - (mv[12] ^ BMW_H[12]) + (mv[15] ^ BMW_H[15]);
	q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + BMW_H[3];
	tmp = (mv[0] ^ BMW_H[0]) - (mv[1] ^ BMW_H[1]) + (mv[8] ^ BMW_H[8]) - (mv[10] ^ BMW_H[10]) + (mv[13] ^ BMW_H[13]);
	q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + BMW_H[4];
	tmp = (mv[1] ^ BMW_H[1]) + (mv[2] ^ BMW_H[2]) + (mv[9] ^ BMW_H[9]) - (mv[11] ^ BMW_H[11]) - (mv[14] ^ BMW_H[14]);
	q[4] = (SHR(tmp, 1) ^ tmp) + BMW_H[5];
	tmp = (mv[3] ^ BMW_H[3]) - (mv[2] ^ BMW_H[2]) + (mv[10] ^ BMW_H[10]) - (mv[12] ^ BMW_H[12]) + (mv[15] ^ BMW_H[15]);
	q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[6];
	tmp = (mv[4] ^ BMW_H[4]) - (mv[0] ^ BMW_H[0]) - (mv[3] ^ BMW_H[3]) - (mv[11] ^ BMW_H[11]) + (mv[13] ^ BMW_H[13]);
	q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + BMW_H[7];
	tmp = (mv[1] ^ BMW_H[1]) - (mv[4] ^ BMW_H[4]) - (mv[5] ^ BMW_H[5]) - (mv[12] ^ BMW_H[12]) - (mv[14] ^ BMW_H[14]);
	q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + BMW_H[8];
	tmp = (mv[2] ^ BMW_H[2]) - (mv[5] ^ BMW_H[5]) - (mv[6] ^ BMW_H[6]) + (mv[13] ^ BMW_H[13]) - (mv[15] ^ BMW_H[15]);
	q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + BMW_H[9];
	tmp = (mv[0] ^ BMW_H[0]) - (mv[3] ^ BMW_H[3]) + (mv[6] ^ BMW_H[6]) - (mv[7] ^ BMW_H[7]) + (mv[14] ^ BMW_H[14]);
	q[9] = (SHR(tmp, 1) ^ tmp) + BMW_H[10];
	tmp = (mv[8] ^ BMW_H[8]) - (mv[1] ^ BMW_H[1]) - (mv[4] ^ BMW_H[4]) - (mv[7] ^ BMW_H[7]) + (mv[15] ^ BMW_H[15]);
	q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[11];
	tmp = (mv[8] ^ BMW_H[8]) - (mv[0] ^ BMW_H[0]) - (mv[2] ^ BMW_H[2]) - (mv[5] ^ BMW_H[5]) + (mv[9] ^ BMW_H[9]);
	q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + BMW_H[12];
	tmp = (mv[1] ^ BMW_H[1]) + (mv[3] ^ BMW_H[3]) - (mv[6] ^ BMW_H[6]) - (mv[9] ^ BMW_H[9]) + (mv[10] ^ BMW_H[10]);
	q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + BMW_H[13];
	tmp = (mv[2] ^ BMW_H[2]) + (mv[4] ^ BMW_H[4]) + (mv[7] ^ BMW_H[7]) + (mv[10] ^ BMW_H[10]) + (mv[11] ^ BMW_H[11]);
	q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + BMW_H[14];
	tmp = (mv[3] ^ BMW_H[3]) - (mv[5] ^ BMW_H[5]) + (mv[8] ^ BMW_H[8]) - (mv[11] ^ BMW_H[11]) - (mv[12] ^ BMW_H[12]);
	q[14] = (SHR(tmp, 1) ^ tmp) + BMW_H[15];
	tmp = (mv[12] ^ BMW_H[12]) - (mv[4] ^ BMW_H[4]) - (mv[6] ^ BMW_H[6]) - (mv[9] ^ BMW_H[9]) + (mv[13] ^ BMW_H[13]);
	q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[0];

	for(i=0;i<2;i++)
	{
		q[i+16] =
		(SHR(q[i], 1) ^ SHL(q[i], 2) ^ ROTL64(q[i], 13) ^ ROTL64(q[i], 43)) +
		(SHR(q[i+1], 2) ^ SHL(q[i+1], 1) ^ ROTL64(q[i+1], 19) ^ ROTL64(q[i+1], 53)) +
		(SHR(q[i+2], 2) ^ SHL(q[i+2], 2) ^ ROTL64(q[i+2], 28) ^ ROTL64(q[i+2], 59)) +
		(SHR(q[i+3], 1) ^ SHL(q[i+3], 3) ^ ROTL64(q[i+3], 4) ^ ROTL64(q[i+3], 37)) +
		(SHR(q[i+4], 1) ^ SHL(q[i+4], 2) ^ ROTL64(q[i+4], 13) ^ ROTL64(q[i+4], 43)) +
		(SHR(q[i+5], 2) ^ SHL(q[i+5], 1) ^ ROTL64(q[i+5], 19) ^ ROTL64(q[i+5], 53)) +
		(SHR(q[i+6], 2) ^ SHL(q[i+6], 2) ^ ROTL64(q[i+6], 28) ^ ROTL64(q[i+6], 59)) +
		(SHR(q[i+7], 1) ^ SHL(q[i+7], 3) ^ ROTL64(q[i+7], 4) ^ ROTL64(q[i+7], 37)) +
		(SHR(q[i+8], 1) ^ SHL(q[i+8], 2) ^ ROTL64(q[i+8], 13) ^ ROTL64(q[i+8], 43)) +
		(SHR(q[i+9], 2) ^ SHL(q[i+9], 1) ^ ROTL64(q[i+9], 19) ^ ROTL64(q[i+9], 53)) +
		(SHR(q[i+10], 2) ^ SHL(q[i+10], 2) ^ ROTL64(q[i+10], 28) ^ ROTL64(q[i+10], 59)) +
		(SHR(q[i+11], 1) ^ SHL(q[i+11], 3) ^ ROTL64(q[i+11], 4) ^ ROTL64(q[i+11], 37)) +
		(SHR(q[i+12], 1) ^ SHL(q[i+12], 2) ^ ROTL64(q[i+12], 13) ^ ROTL64(q[i+12], 43)) +
		(SHR(q[i+13], 2) ^ SHL(q[i+13], 1) ^ ROTL64(q[i+13], 19) ^ ROTL64(q[i+13], 53)) +
		(SHR(q[i+14], 2) ^ SHL(q[i+14], 2) ^ ROTL64(q[i+14], 28) ^ ROTL64(q[i+14], 59)) +
		(SHR(q[i+15], 1) ^ SHL(q[i+15], 3) ^ ROTL64(q[i+15], 4) ^ ROTL64(q[i+15], 37)) +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i+10], i+11) ) ^ BMW_H[i+7]);
	}

	for(i=2;i<6;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i+10], i+11) ) ^ BMW_H[i+7]);
	}

	for(i=6;i<9;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i-6], (i-6)+1) ) ^ BMW_H[i+7]);
	}

	for(i=9;i<13;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i-6], (i-6)+1) ) ^ BMW_H[i-9]);
	}

	for(i=13;i<16;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i-13], (i-13)+1) - ROTL64(mv[i-6], (i-6)+1) ) ^ BMW_H[i-9]);
	}

	uint64 XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	uint64 XH64 = XL64^q[24]^q[25]^q[26]^q[27]^q[28]^q[29]^q[30]^q[31];

	BMW_H[0] = (SHL(XH64, 5) ^ SHR(q[16],5) ^ mv[0]) + ( XL64 ^ q[24] ^ q[0]);
	BMW_H[1] = (SHR(XH64, 7) ^ SHL(q[17],8) ^ mv[1]) + ( XL64 ^ q[25] ^ q[1]);
	BMW_H[2] = (SHR(XH64, 5) ^ SHL(q[18],5) ^ mv[2]) + ( XL64 ^ q[26] ^ q[2]);
	BMW_H[3] = (SHR(XH64, 1) ^ SHL(q[19],5) ^ mv[3]) + ( XL64 ^ q[27] ^ q[3]);
	BMW_H[4] = (SHR(XH64, 3) ^ q[20] ^ mv[4]) + ( XL64 ^ q[28] ^ q[4]);
	BMW_H[5] = (SHL(XH64, 6) ^ SHR(q[21],6) ^ mv[5]) + ( XL64 ^ q[29] ^ q[5]);
	BMW_H[6] = (SHR(XH64, 4) ^ SHL(q[22],6) ^ mv[6]) + ( XL64 ^ q[30] ^ q[6]);
	BMW_H[7] = (SHR(XH64,11) ^ SHL(q[23],2) ^ mv[7]) + ( XL64 ^ q[31] ^ q[7]);

	BMW_H[8] = ROTL64(BMW_H[4], 9) + ( XH64 ^ q[24] ^ mv[8]) + (SHL(XL64,8) ^ q[23] ^ q[8]);
	BMW_H[9] = ROTL64(BMW_H[5],10) + ( XH64 ^ q[25] ^ mv[9]) + (SHR(XL64,6) ^ q[16] ^ q[9]);
	BMW_H[10] = ROTL64(BMW_H[6],11) + ( XH64 ^ q[26] ^ mv[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
	BMW_H[11] = ROTL64(BMW_H[7],12) + ( XH64 ^ q[27] ^ mv[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
	BMW_H[12] = ROTL64(BMW_H[0],13) + ( XH64 ^ q[28] ^ mv[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
	BMW_H[13] = ROTL64(BMW_H[1],14) + ( XH64 ^ q[29] ^ mv[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
	BMW_H[14] = ROTL64(BMW_H[2],15) + ( XH64 ^ q[30] ^ mv[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	BMW_H[15] = ROTL64(BMW_H[3],16) + ( XH64 ^ q[31] ^ mv[15]) + (SHR(XL64,2) ^ q[22] ^ q[15]);

	/* final */
	for(i=0;i<16;i++)
	{
		mv[i] = BMW_H[i];
		BMW_H[i] = 0xaaaaaaaaaaaaaaa0ul + (uint64)i;
	}

	tmp = (mv[5] ^ BMW_H[5]) - (mv[7] ^ BMW_H[7]) + (mv[10] ^ BMW_H[10]) + (mv[13] ^ BMW_H[13]) + (mv[14] ^ BMW_H[14]);
	q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[1];
	tmp = (mv[6] ^ BMW_H[6]) - (mv[8] ^ BMW_H[8]) + (mv[11] ^ BMW_H[11]) + (mv[14] ^ BMW_H[14]) - (mv[15] ^ BMW_H[15]);
	q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + BMW_H[2];
	tmp = (mv[0] ^ BMW_H[0]) + (mv[7] ^ BMW_H[7]) + (mv[9] ^ BMW_H[9]) - (mv[12] ^ BMW_H[12]) + (mv[15] ^ BMW_H[15]);
	q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + BMW_H[3];
	tmp = (mv[0] ^ BMW_H[0]) - (mv[1] ^ BMW_H[1]) + (mv[8] ^ BMW_H[8]) - (mv[10] ^ BMW_H[10]) + (mv[13] ^ BMW_H[13]);
	q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + BMW_H[4];
	tmp = (mv[1] ^ BMW_H[1]) + (mv[2] ^ BMW_H[2]) + (mv[9] ^ BMW_H[9]) - (mv[11] ^ BMW_H[11]) - (mv[14] ^ BMW_H[14]);
	q[4] = (SHR(tmp, 1) ^ tmp) + BMW_H[5];
	tmp = (mv[3] ^ BMW_H[3]) - (mv[2] ^ BMW_H[2]) + (mv[10] ^ BMW_H[10]) - (mv[12] ^ BMW_H[12]) + (mv[15] ^ BMW_H[15]);
	q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[6];
	tmp = (mv[4] ^ BMW_H[4]) - (mv[0] ^ BMW_H[0]) - (mv[3] ^ BMW_H[3]) - (mv[11] ^ BMW_H[11]) + (mv[13] ^ BMW_H[13]);
	q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + BMW_H[7];
	tmp = (mv[1] ^ BMW_H[1]) - (mv[4] ^ BMW_H[4]) - (mv[5] ^ BMW_H[5]) - (mv[12] ^ BMW_H[12]) - (mv[14] ^ BMW_H[14]);
	q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + BMW_H[8];
	tmp = (mv[2] ^ BMW_H[2]) - (mv[5] ^ BMW_H[5]) - (mv[6] ^ BMW_H[6]) + (mv[13] ^ BMW_H[13]) - (mv[15] ^ BMW_H[15]);
	q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + BMW_H[9];
	tmp = (mv[0] ^ BMW_H[0]) - (mv[3] ^ BMW_H[3]) + (mv[6] ^ BMW_H[6]) - (mv[7] ^ BMW_H[7]) + (mv[14] ^ BMW_H[14]);
	q[9] = (SHR(tmp, 1) ^ tmp) + BMW_H[10];
	tmp = (mv[8] ^ BMW_H[8]) - (mv[1] ^ BMW_H[1]) - (mv[4] ^ BMW_H[4]) - (mv[7] ^ BMW_H[7]) + (mv[15] ^ BMW_H[15]);
	q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[11];
	tmp = (mv[8] ^ BMW_H[8]) - (mv[0] ^ BMW_H[0]) - (mv[2] ^ BMW_H[2]) - (mv[5] ^ BMW_H[5]) + (mv[9] ^ BMW_H[9]);
	q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + BMW_H[12];
	tmp = (mv[1] ^ BMW_H[1]) + (mv[3] ^ BMW_H[3]) - (mv[6] ^ BMW_H[6]) - (mv[9] ^ BMW_H[9]) + (mv[10] ^ BMW_H[10]);
	q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + BMW_H[13];
	tmp = (mv[2] ^ BMW_H[2]) + (mv[4] ^ BMW_H[4]) + (mv[7] ^ BMW_H[7]) + (mv[10] ^ BMW_H[10]) + (mv[11] ^ BMW_H[11]);
	q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + BMW_H[14];
	tmp = (mv[3] ^ BMW_H[3]) - (mv[5] ^ BMW_H[5]) + (mv[8] ^ BMW_H[8]) - (mv[11] ^ BMW_H[11]) - (mv[12] ^ BMW_H[12]);
	q[14] = (SHR(tmp, 1) ^ tmp) + BMW_H[15];
	tmp = (mv[12] ^ BMW_H[12]) - (mv[4] ^ BMW_H[4]) - (mv[6] ^ BMW_H[6]) - (mv[9] ^ BMW_H[9]) + (mv[13] ^ BMW_H[13]);
	q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + BMW_H[0];

	for(i=0;i<2;i++)
	{
		q[i+16] =
		(SHR(q[i], 1) ^ SHL(q[i], 2) ^ ROTL64(q[i], 13) ^ ROTL64(q[i], 43)) +
		(SHR(q[i+1], 2) ^ SHL(q[i+1], 1) ^ ROTL64(q[i+1], 19) ^ ROTL64(q[i+1], 53)) +
		(SHR(q[i+2], 2) ^ SHL(q[i+2], 2) ^ ROTL64(q[i+2], 28) ^ ROTL64(q[i+2], 59)) +
		(SHR(q[i+3], 1) ^ SHL(q[i+3], 3) ^ ROTL64(q[i+3], 4) ^ ROTL64(q[i+3], 37)) +
		(SHR(q[i+4], 1) ^ SHL(q[i+4], 2) ^ ROTL64(q[i+4], 13) ^ ROTL64(q[i+4], 43)) +
		(SHR(q[i+5], 2) ^ SHL(q[i+5], 1) ^ ROTL64(q[i+5], 19) ^ ROTL64(q[i+5], 53)) +
		(SHR(q[i+6], 2) ^ SHL(q[i+6], 2) ^ ROTL64(q[i+6], 28) ^ ROTL64(q[i+6], 59)) +
		(SHR(q[i+7], 1) ^ SHL(q[i+7], 3) ^ ROTL64(q[i+7], 4) ^ ROTL64(q[i+7], 37)) +
		(SHR(q[i+8], 1) ^ SHL(q[i+8], 2) ^ ROTL64(q[i+8], 13) ^ ROTL64(q[i+8], 43)) +
		(SHR(q[i+9], 2) ^ SHL(q[i+9], 1) ^ ROTL64(q[i+9], 19) ^ ROTL64(q[i+9], 53)) +
		(SHR(q[i+10], 2) ^ SHL(q[i+10], 2) ^ ROTL64(q[i+10], 28) ^ ROTL64(q[i+10], 59)) +
		(SHR(q[i+11], 1) ^ SHL(q[i+11], 3) ^ ROTL64(q[i+11], 4) ^ ROTL64(q[i+11], 37)) +
		(SHR(q[i+12], 1) ^ SHL(q[i+12], 2) ^ ROTL64(q[i+12], 13) ^ ROTL64(q[i+12], 43)) +
		(SHR(q[i+13], 2) ^ SHL(q[i+13], 1) ^ ROTL64(q[i+13], 19) ^ ROTL64(q[i+13], 53)) +
		(SHR(q[i+14], 2) ^ SHL(q[i+14], 2) ^ ROTL64(q[i+14], 28) ^ ROTL64(q[i+14], 59)) +
		(SHR(q[i+15], 1) ^ SHL(q[i+15], 3) ^ ROTL64(q[i+15], 4) ^ ROTL64(q[i+15], 37)) +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i+10], i+11) ) ^ BMW_H[i+7]);
	}

	for(i=2;i<6;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i+10], i+11) ) ^ BMW_H[i+7]);
	}

	for(i=6;i<9;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i-6], (i-6)+1) ) ^ BMW_H[i+7]);
	}

	for(i=9;i<13;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i+3], i+4) - ROTL64(mv[i-6], (i-6)+1) ) ^ BMW_H[i-9]);
	}

	for(i=13;i<16;i++)
	{
		q[i+16] = CONST_EXP2 +
		(( ((i+16)*(0x0555555555555555ul)) + ROTL64(mv[i], i+1) +
		ROTL64(mv[i-13], (i-13)+1) - ROTL64(mv[i-6], (i-6)+1) ) ^ BMW_H[i-9]);
	}

	XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	XH64 = XL64^q[24]^q[25]^q[26]^q[27]^q[28]^q[29]^q[30]^q[31];

	BMW_H[0] = (SHL(XH64, 5) ^ SHR(q[16],5) ^ mv[0]) + ( XL64 ^ q[24] ^ q[0]);
	BMW_H[1] = (SHR(XH64, 7) ^ SHL(q[17],8) ^ mv[1]) + ( XL64 ^ q[25] ^ q[1]);
	BMW_H[2] = (SHR(XH64, 5) ^ SHL(q[18],5) ^ mv[2]) + ( XL64 ^ q[26] ^ q[2]);
	BMW_H[3] = (SHR(XH64, 1) ^ SHL(q[19],5) ^ mv[3]) + ( XL64 ^ q[27] ^ q[3]);
	BMW_H[4] = (SHR(XH64, 3) ^ q[20] ^ mv[4]) + ( XL64 ^ q[28] ^ q[4]);
	BMW_H[5] = (SHL(XH64, 6) ^ SHR(q[21],6) ^ mv[5]) + ( XL64 ^ q[29] ^ q[5]);
	BMW_H[6] = (SHR(XH64, 4) ^ SHL(q[22],6) ^ mv[6]) + ( XL64 ^ q[30] ^ q[6]);
	BMW_H[7] = (SHR(XH64,11) ^ SHL(q[23],2) ^ mv[7]) + ( XL64 ^ q[31] ^ q[7]);

	BMW_H[8] = ROTL64(BMW_H[4], 9) + ( XH64 ^ q[24] ^ mv[8]) + (SHL(XL64,8) ^ q[23] ^ q[8]);
	BMW_H[9] = ROTL64(BMW_H[5],10) + ( XH64 ^ q[25] ^ mv[9]) + (SHR(XL64,6) ^ q[16] ^ q[9]);
	BMW_H[10] = ROTL64(BMW_H[6],11) + ( XH64 ^ q[26] ^ mv[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
	BMW_H[11] = ROTL64(BMW_H[7],12) + ( XH64 ^ q[27] ^ mv[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
	BMW_H[12] = ROTL64(BMW_H[0],13) + ( XH64 ^ q[28] ^ mv[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
	BMW_H[13] = ROTL64(BMW_H[1],14) + ( XH64 ^ q[29] ^ mv[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
	BMW_H[14] = ROTL64(BMW_H[2],15) + ( XH64 ^ q[30] ^ mv[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	BMW_H[15] = ROTL64(BMW_H[3],16) + ( XH64 ^ q[31] ^ mv[15]) + (SHR(XL64,2) ^ q[22] ^ q[15]);

	enc64le2(hashes + (tid*64) + 0, BMW_H[8]);
	enc64le2(hashes + (tid*64) + 8, BMW_H[9]);
	enc64le2(hashes + (tid*64) + 16, BMW_H[10]);
	enc64le2(hashes + (tid*64) + 24, BMW_H[11]);
	enc64le2(hashes + (tid*64) + 32, BMW_H[12]);
	enc64le2(hashes + (tid*64) + 40, BMW_H[13]);
	enc64le2(hashes + (tid*64) + 48, BMW_H[14]);
	enc64le2(hashes + (tid*64) + 56, BMW_H[15]);

	//__syncthreads();
}